#include "hip/hip_runtime.h"
// slope_aspect.cu  (compiled with nvcc -O3 --use_fast_math)
__global__ void slopeAspect(const float* dem, float* slope,
                            float* aspect, int W, int H, float cellSz)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;  // pixel coords
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x <= 0 || y <= 0 || x >= W-1 || y >= H-1) return;

    int idx = y * W + x;
    float dzdx = (dem[idx+1] - dem[idx-1]) / (2*cellSz);
    float dzdy = (dem[idx+W] - dem[idx-W]) / (2*cellSz);
    slope [idx] = atan(sqrtf(dzdx*dzdx + dzdy*dzdy));
    aspect[idx] = atan2f(-dzdy, dzdx);
}
hipGraph_t g; hipStream_t s;
hipStreamCreate(&s);
hipStreamBeginCapture(s, hipStreamCaptureModeGlobal);
dim3 blk(32,32), grid((W+31)/32,(H+31)/32);
slopeAspect<<<grid, blk, 0, s>>>(dDEM, dSlope, dAspect, W, H, cell);
hipStreamEndCapture(s, &g);
hipGraphExec_t gExec; hipGraphInstantiate(&gExec, g, nullptr, nullptr, 0);

for (auto roi : regions) {                   // every time user selects a ROI
    hipMemcpyAsync(dDEM, roi.data, bytes, hipMemcpyHostToDevice, s);
    hipGraphLaunch(gExec, s);               // launches whole chain once
    hipMemcpyAsync(out.data, dSlope, bytes, hipMemcpyDeviceToHost, s);
}
